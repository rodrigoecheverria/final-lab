#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <hip/hip_runtime.h>
#include <math.h>

#define TILE_DIM 4

struct plusOneFunc {
        __host__ __device__ float operator()(float z) const {
        return z+1;
    }
};
template<typename UnaryFunction>
__global__ void Tranform(float* Y, int Rows, int Cols, UnaryFunction activationFunction)
{
    int Row = blockIdx.y * TILE_DIM + threadIdx.y;
    int Col = blockIdx.x * TILE_DIM + threadIdx.x;
    
    if (Row < Rows && Col < Cols) 
        Y[((blockIdx.y * blockDim.y + threadIdx.y) * Cols) + 
            (blockIdx.x * blockDim.x) + threadIdx.x] = 
            activationFunction(Y[((blockIdx.y * blockDim.y + threadIdx.y) * Cols) + 
            (blockIdx.x * blockDim.x) + threadIdx.x]);
}

// Invoke kernel
int main(int argc, char *argv[])
{
    float *d_A, *d_B, *d_C, *A, *B, *C;
    int i, N =6, M = 6;
    A = (float *) malloc (sizeof(float) * M * N);
    B = (float *) malloc (sizeof(float) * M * N);
    C = (float *) malloc (sizeof(float) * M * M);
    hipMalloc((void **)&d_A, M * N * sizeof(float));
    hipMalloc((void **)&d_B, M * N * sizeof(float));
    hipMalloc((void **)&d_C, M * M * sizeof(float));
    
    for (i = 0; i < N * M; i++) A[i] =i;
    for (i = 0; i < M * N; i++) B[i] = i;
    for (i = 0; i < M * M; i++) C[i] = 0.0;
	
    hipMemcpy(d_A,	A, N * M * sizeof(float), hipMemcpyHostToDevice);	
    hipMemcpy(d_B,	B, M * N * sizeof(float), hipMemcpyHostToDevice);
    
    dim3 dimBlock(TILE_DIM, TILE_DIM);
    dim3 dimGrid((N + dimBlock.x -1) / dimBlock.x, (M  + dimBlock.y -1) / dimBlock.y);
    plusOneFunc sigmoidf;
		Tranform<<<dimGrid, dimBlock>>>(d_A,N,M,sigmoidf);
    hipDeviceSynchronize();
    
    hipMemcpy(A, d_A, N * M * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(d_A); hipFree(d_B); hipFree(d_C);
    
    for (i = 0; i < N * M; i++)
    {
			printf("%f, ",A[i]);
			printf("\n");
    }
    return 0;
    
}
