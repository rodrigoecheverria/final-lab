#include "hip/hip_runtime.h"
 __global__ void Sigmoid(float* A, int ARows, int ACols) {

    int Col = blockIdx.x*BLOCK_SIZE + threadIdx.x;
    int Row = blockIdx.y*BLOCK_SIZE + threadIdx.y;
    

    for (int k = 0; k < (TILE_DIM + ACols - 1)/TILE_DIM; k++) {

        for (int n = 0; n < TILE_DIM; ++n) 
            if ((k*TILE_DIM + n < ACols && Row < ARows) && (k*TILE_DIM + n < BRows && Col < BCols))
                CValue += A[Row*ACols + k*TILE_DIM + n] * B[(k*TILE_DIM + n)*BCols + Col];

    }

    if (Row < CRows && Col < CCols) C[((blockIdx.y * blockDim.y + threadIdx.y)*CCols)+(blockIdx.x*blockDim.x)+threadIdx.x]=CValue;
}
