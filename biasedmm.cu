#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <hip/hip_runtime.h>
#include <math.h>

#define TILE_DIM 4

struct sigmoidFunc {
        __host__ __device__ float operator()(float z) const {
        return 1.0/(1.0 + exp(-(z/1000)));
    }
};  

//TRY TO PUT A SIGMOID FUNCTOR HERE !!!!
template<typename UnaryFunction>
__global__ void MatMul(float* A, float* B, float* C, int ARows, int ACols, 
    int BRows, int BCols, int CRows, int CCols, bool addBias, UnaryFunction activationFunction ) 
{
    float CValue = 0;
    int Row = blockIdx.y * TILE_DIM + threadIdx.y;
    int Col = blockIdx.x * TILE_DIM + threadIdx.x;
    int biasOffset = addBias ? 1 : 0; 
	
    __shared__ float As[TILE_DIM][TILE_DIM];
    __shared__ float Bs[TILE_DIM][TILE_DIM];

    for (int k = 0; k < (TILE_DIM + ACols - 1)/TILE_DIM; k++)           //floor(ACols/TILE_DIM)
    {
         if (k * TILE_DIM + threadIdx.x < ACols && Row < ARows)   
            As[threadIdx.y][threadIdx.x] = A[Row * ACols + k * TILE_DIM + threadIdx.x];
         else                                                 
            As[threadIdx.y][threadIdx.x] = 0.0;

         if (k * TILE_DIM + threadIdx.y < BRows && Col < BCols)   
            Bs[threadIdx.y][threadIdx.x] = B[(k * TILE_DIM + threadIdx.y + biasOffset) * BCols + Col]; //+1 one row if bias
         else      
            Bs[threadIdx.y][threadIdx.x] = 0.0;

         __syncthreads();

         for (int n = 0; n < TILE_DIM; ++n) 
            CValue += As[threadIdx.y][n] * Bs[n][threadIdx.x];

         __syncthreads();
    }
    if (addBias)
	{
		__shared__ float BiasRow[TILE_DIM];
	    
		if (threadIdx.y == 0){
		  if (Col < BCols){
			BiasRow[threadIdx.x] = B[Col];
		  }else{
		  	BiasRow[threadIdx.x] = 0.0;
			}
			}
	    __syncthreads();
		
		CValue += BiasRow[threadIdx.x];
		
		__syncthreads();
		
	}
	
    if (Row < CRows && Col < CCols) 
        C[((blockIdx.y * blockDim.y + threadIdx.y) * CCols) + 
            (blockIdx.x * blockDim.x) + threadIdx.x] = 
            activationFunction(CValue);
}


// Invoke kernel
int main(int argc, char *argv[])
{
    float *d_A, *d_B, *d_C, *A, *B, *C;
    int i, N =6, M = 3;
    A = (float *) malloc (sizeof(float) * M * N);
    B = (float *) malloc (sizeof(float) * M * (N+1));
    C = (float *) malloc (sizeof(float) * M * M);
    hipMalloc((void **)&d_A, M * N * sizeof(float));
    hipMalloc((void **)&d_B, M * (N+1) * sizeof(float));
    hipMalloc((void **)&d_C, M * M * sizeof(float));
    
    for (i = 0; i < N * M; i++) A[i] =i;
    for (i = 0; i < M * (N+1); i++) B[i] = i;
    for (i = 0; i < M * M; i++) C[i] = 0.0;
	
    hipMemcpy(d_A,	A, N * M * sizeof(float), hipMemcpyHostToDevice);	
    hipMemcpy(d_B,	B, M * (N + 1) * sizeof(float), hipMemcpyHostToDevice);
    
    dim3 dimBlock(TILE_DIM, TILE_DIM);
    dim3 dimGrid((M + dimBlock.x -1) / dimBlock.x, (M  + dimBlock.y -1) / dimBlock.y);
    sigmoidFunc sigmoidf;
		MatMul<<<dimGrid, dimBlock>>>(d_A, d_B, d_C,M,N,N,M,M,M,true,sigmoidf);
    hipDeviceSynchronize();
    
    hipMemcpy(C, d_C, M * M * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(d_A); hipFree(d_B); hipFree(d_C);
    
    for (i = 0; i < M * M; i++)
    {
			if ((i % N) == 0) printf("\n");
			printf("%f, ",C[i]);
    }
    return 0;
    
}
