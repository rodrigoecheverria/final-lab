#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#define TILE_DIM 16

__global__ void MatMul(float* A, float* B, float* C, int ARows, int ACols, 
    int BRows, int BCols, int CRows, int CCols) 
{
    float CValue = 0;
    int Row = blockIdx.y * TILE_DIM + threadIdx.y;
    int Col = blockIdx.x * TILE_DIM + threadIdx.x;

    __shared__ float As[TILE_DIM][TILE_DIM];
    __shared__ float Bs[TILE_DIM][TILE_DIM];

    for (int k = 0; k < (TILE_DIM + ACols - 1)/TILE_DIM; k++)
    {
         if (k * TILE_DIM + threadIdx.x < ACols && Row < ARows)   
            As[threadIdx.y][threadIdx.x] = 
                A[Row * ACols + k * TILE_DIM + threadIdx.x];
         else                                                 
            As[threadIdx.y][threadIdx.x] = 0.0;

         if (k * TILE_DIM + threadIdx.y < BRows && Col < BCols)   
            Bs[threadIdx.y][threadIdx.x] = 
                B[(k * TILE_DIM + threadIdx.y) * BCols + Col];
         else      
            Bs[threadIdx.y][threadIdx.x] = 0.0;

         __syncthreads();

         for (int n = 0; n < TILE_DIM; ++n) 
            CValue += As[threadIdx.y][n] * Bs[n][threadIdx.x];

         __syncthreads();
    }

    if (Row < CRows && Col < CCols) 
        C[((blockIdx.y * blockDim.y + threadIdx.y) * CCols) + 
            (blockIdx.x * blockDim.x) + threadIdx.x] = CValue;
}


// Invoke kernel
int main(int argc, char *argv[])
{
    float *d_A, *d_B, *d_C, *A, *B, *C;
    int i, N = 32;
    A = (float *) malloc (sizeof(float) * N * N);
    B = (float *) malloc (sizeof(float) * N * N);
    C = (float *) malloc (sizeof(float) * N * N);
    hipMalloc((void **)&d_A, N * N * sizeof(float));
    hipMalloc((void **)&d_B, N * N * sizeof(float));
    hipMalloc((void **)&d_C, N * N * sizeof(float));
    
    for (i = 0; i < N * N; i++)
    {
        A[i] = 1.0; B[i] = 1.0; C[i] = 0.0;
    }
    
    hipMemcpy(d_A,	A, N * N * sizeof(float), hipMemcpyHostToDevice);	
    hipMemcpy(d_B,	B, N * N * sizeof(float), hipMemcpyHostToDevice);
    
    dim3 dimBlock(TILE_DIM, TILE_DIM);
    dim3 dimGrid(N / dimBlock.x, N / dimBlock.y);
    MatMul<<<dimGrid, dimBlock>>>(d_A, d_B, d_C,N,N,N,N,N,N);
    hipDeviceSynchronize();
    
    hipMemcpy(C, d_C, N * N * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(d_A); hipFree(d_B); hipFree(d_C);
    
    for (i = 0; i < N * N; i++)
    {
        printf("%f, ",C[i]);
    }
    return 0;
    
}
