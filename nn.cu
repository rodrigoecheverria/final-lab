#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <tclap/CmdLine.h> //-I wherever TCLAP is
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <math.h>

#define TILE_DIM 8
//========================================================================================
// SIGNATURES
//========================================================================================
struct sigmoidFunc {
        __host__ __device__ float operator()(float z) const {
        return 1.0/(1.0 + exp(-(z)));
    }
};

struct plusFunc {
        __host__ __device__ float operator()(float x, float y) const {
        return x + y;
    }
};

struct prodFunc {
        __host__ __device__ float operator()(float x, float y) const {
        return x * y;
    }
};

struct goodLogisticRegressionFunc {
        __host__ __device__ float operator()(float x, float y) const {
        return (-x * log(y));
    }
};

struct badLogisticRegressionFunc {
        __host__ __device__ float operator()(float x, float y) const {
        return ((1-x) * log(1-y));
    }
};

template<typename UnaryFunction>
__global__ void MatMul(float* A, float* B, float* C, int ARows, int ACols, int BRows, 
                       int BCols, int CRows, int CCols, bool addBias, 
                       UnaryFunction activationFunction);
template<typename MapFunction,
         typename ReduceFunction>                       
__global__ void ZipMapReduceKernel(float* X, float* Y, float* R, int size, 
                                    MapFunction mapFunction, float neutralElement, 
                                    ReduceFunction reduceFunction);
template<typename MapFunction,
         typename ReduceFunction>                                                           
float ZipMapReduce(float* d_X, float* d_Y, int size, MapFunction mapFunction, 
                   float neutralElement, ReduceFunction reduceFunction);
                   
std::vector<int>& splitToInts(const std::string &s, char delim, std::vector<int> &elems);
std::vector<int> splitToInts(const std::string &s, char delim);

struct Options 
{
    int numberOfLayers;
	std::vector<int> layerSizes;
	std::string activationFunction;
	std::string samplesFile;
	std::string resultsFile;
	int numberOfTrainingSamples;
};
struct Options ParseCommandLine(int argc, char *argv[]);
void readCsvIntoMatrix(const std::string fileName, float* M, const int rows, 
                        const int columns);
void readResultsIntoMatrix(const std::string fileName, float* M, const int rows, 
                            const int columns);
void GPU_fill_rand(float *A, int nr_rows_A, int nr_cols_A);
void printMatrix(float *M, int rows, int columns);

int main(int argc, char *argv[])
{
    int i,l;
    float *X, *Y, *d_Y, **Theta, **d_Theta, **a, **d_a, J; 
    //d_Theta and d_a are host vectors of pointers to device memory!
    hipError_t err;
    sigmoidFunc sigmoidf;
    goodLogisticRegressionFunc goodLogisticRegressionf;
    badLogisticRegressionFunc badLogisticRegressionf;
    plusFunc plusf;
    prodFunc prodf;
    
	Options options = ParseCommandLine(argc,argv);
	
	printf("Number of layers: %d\n", options.numberOfLayers);
	
	printf("Activation function: %s\n", options.activationFunction.c_str());
	//==========================================================================
	// Allocate memory in both host and device
	//==========================================================================
	X = (float *) malloc (sizeof(float) * options.numberOfTrainingSamples * 
	                        options.layerSizes[0]);
	Y = (float *) malloc (sizeof(float) * options.numberOfTrainingSamples *
	                        options.layerSizes.back());
	Theta = (float **) malloc ((options.numberOfLayers - 1) * sizeof(float*)); 
	d_Theta = (float **) malloc ((options.numberOfLayers - 1) * sizeof(float*));
	a = (float **) malloc (options.numberOfLayers * sizeof(float*));
	d_a = (float **) malloc (options.numberOfLayers * sizeof(float*));
	
    hipMalloc((void **) &d_Y, options.numberOfTrainingSamples * 
                options.layerSizes.back() * sizeof(float));
	for (i = 0; i < options.numberOfLayers - 1; i++)
	{
	    Theta[i] =  (float *) malloc (sizeof(float) * (options.layerSizes[i] + 1) * 
	                    options.layerSizes[i+1]); //+1 is the bias row
	    if (Theta[i] == NULL)
	        printf ("MALLOC ERROR\n");                
        err = hipMalloc((void **)&(d_Theta[i]), (options.layerSizes[i] + 1) * 
                options.layerSizes[i+1] * sizeof(float)); //+1 is the bias row
        if (err > 0) printf("error code: %d\n",err);
    }
    
    for (l = 0; l < options.numberOfLayers; l++) 
    {
        //an activation for each training sample per each neuron at layer l
        err = hipMalloc((void **)&(d_a[l]), options.layerSizes[l] * 
                options.numberOfTrainingSamples * sizeof(float)); 
        a[l] = (float *) malloc(sizeof(float) * options.layerSizes[l] * 
                options.numberOfTrainingSamples); 
        if (err > 0) printf("error code: %d\n",err);
        
    }
    
    
	//==========================================================================
	// INITIALIZE VALUES
	//==========================================================================
	readCsvIntoMatrix(options.samplesFile, X, options.numberOfTrainingSamples, 
                        options.layerSizes[0]);
	readResultsIntoMatrix(options.resultsFile, Y, options.numberOfTrainingSamples, 
                            options.layerSizes.back());
    //for (i = 0; i < options.numberOfLayers - 1; i++)
    //    GPU_fill_rand(d_Theta[i], options.layerSizes[i] + 1, options.layerSizes[i+1]);
	readCsvIntoMatrix("data_theta0.csv", Theta[0], options.layerSizes[0] + 1, 
                        options.layerSizes[1]);
	readCsvIntoMatrix("data_theta1.csv", Theta[1], options.layerSizes[1] + 1, 
                        options.layerSizes[2]);
    hipMemcpy(d_Theta[0], Theta[0], (options.layerSizes[0] + 1) * options.layerSizes[1] * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_Theta[1], Theta[1], (options.layerSizes[1] + 1) * options.layerSizes[2] * sizeof(float), hipMemcpyHostToDevice);
                        
	//==========================================================================
	// COMPUTE
	//==========================================================================
	printf ("Computing ----------------------------------------------------\n");
	//Feed the X to the first activation function: d_a[0]
	//Fedd the Y
	hipMemcpy(d_a[0], X, options.numberOfTrainingSamples * 
	            options.layerSizes[0] * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_Y, Y, options.numberOfTrainingSamples * 
	           options.layerSizes.back() * sizeof(float), hipMemcpyHostToDevice);
	           
	dim3 dimBlock(TILE_DIM, TILE_DIM);
	// d_a[1] = d_a[0] x d_Theta[0]
	//multiplication scope, so I can reuse (redeclare) dimGrid ;)
	{ 
    dim3 dimGrid((options.layerSizes[1] + dimBlock.x - 1)/ dimBlock.x, 
                 (options.numberOfTrainingSamples + dimBlock.y - 1)/ dimBlock.y);
                 
    MatMul<<<dimGrid, dimBlock>>>(d_a[0], d_Theta[0], d_a[1],
                                    options.numberOfTrainingSamples,
                                    options.layerSizes[0],
                                    options.layerSizes[0],
                                    options.layerSizes[1],
                                    options.numberOfTrainingSamples,
                                    options.layerSizes[1],
                                    true, sigmoidf);
    hipDeviceSynchronize();
    }
    {
    dim3 dimGrid((options.layerSizes[2] + dimBlock.x - 1) / dimBlock.x, 
                 (options.numberOfTrainingSamples + dimBlock.y - 1)/ dimBlock.y);
                 
    MatMul<<<dimGrid, dimBlock>>>(d_a[1], d_Theta[1], d_a[2],
                                    options.numberOfTrainingSamples,
                                    options.layerSizes[1],
                                    options.layerSizes[1],
                                    options.layerSizes[2],
                                    options.numberOfTrainingSamples,
                                    options.layerSizes[2],
                                    true, sigmoidf);
    hipDeviceSynchronize();
    }
    
	
	
    //hipMemcpy(Theta[0], d_Theta[0], (options.layerSizes[0] + 1) * options.layerSizes[1] * sizeof(float), hipMemcpyDeviceToHost);
    //hipMemcpy(Theta[1], d_Theta[1], (options.layerSizes[1] + 1) * options.layerSizes[2] * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(a[0], d_a[0], options.numberOfTrainingSamples * options.layerSizes[0] * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(a[1], d_a[1], options.numberOfTrainingSamples * options.layerSizes[1] * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(a[2], d_a[2], options.numberOfTrainingSamples * options.layerSizes[2] * sizeof(float), hipMemcpyDeviceToHost);
    
	//printMatrix(a[0], options.numberOfTrainingSamples, options.layerSizes[0]);
	//printMatrix(Theta[0], options.layerSizes[0]+1, options.layerSizes[1]);
	//printMatrix(a[1], options.numberOfTrainingSamples, options.layerSizes[1]);
	//printMatrix(Theta[1], options.layerSizes[1]+1, options.layerSizes[2]);
	//printMatrix(a[2], options.numberOfTrainingSamples, options.layerSizes[2]);
	//printMatrix(Y,options.numberOfTrainingSamples,options.layerSizes.back());
	
	//Cost
	J = ZipMapReduce(d_Y, d_a[options.numberOfLayers - 1], 
	                 options.numberOfTrainingSamples * options.layerSizes.back(),        
	                 goodLogisticRegressionf, 0.0, plusf) -
	    ZipMapReduce(d_Y, d_a[options.numberOfLayers - 1], 
	                 options.numberOfTrainingSamples * options.layerSizes.back(), 
	                 badLogisticRegressionf,0.0,plusf);
	              
	J = J / options.numberOfTrainingSamples; //Average
	
	printf("Cost: %f\n",J); 
    
    //Regularized cost
	float coef = 0.0;
	for (i = 0; i < options.numberOfLayers - 1; i++)
	    coef += ZipMapReduce(d_Theta[i]+options.layerSizes[i+1], 
	                         d_Theta[i]+options.layerSizes[i+1],
	                          options.layerSizes[i] * options.layerSizes[i+1],                
	                          prodf,0.0,plusf);
	                          
	J += ( coef /(2*options.numberOfTrainingSamples));
	printf("Coef: %f\n",coef);
	printf("Regularized cost: %f\n",J); 
	
	
	//    hipFree(d_X); hipFree(d_B); hipFree(d_C);    
	return 0;
}


///
/// Helper Functions
///
struct Options ParseCommandLine(int argc, char *argv[])
{
    struct Options options;
    TCLAP::CmdLine cmd("Command description message", ' ', "0.9");
	
	TCLAP::ValueArg<int> numLayersArg("L", "number-of-layers", 
	    "Number of layers considering the input and output layers", false, 3, 
	    "integer", cmd);
	    
	TCLAP::ValueArg<int> numTrainingSamplesArg("T", "number-training-samples", 
	    "Number of training samples to use", true, 0, "integer", cmd);
	    
	TCLAP::ValueArg<std::string> layersArg("l", "layer-sizes", 
	    "Number of neurons for each layer", true, "", "list of integers",cmd);
	
	std::vector<std::string> allowedActivationFunctions;
		allowedActivationFunctions.push_back("sigmoid");
		allowedActivationFunctions.push_back("htan");
	TCLAP::ValuesConstraint<std::string> allowedValsActivationFunction( 
	    allowedActivationFunctions );
	
	TCLAP::ValueArg<std::string> activationFunctionArg("a",
	    "activation-function", "Activation function", false, "sigmoid", 
	    &allowedValsActivationFunction, cmd);
	    
	TCLAP::ValueArg<std::string> fileXArg ("x", "samples", 
	    "File containing the training examples", true, "", 
	    "file name or path", cmd);
	    
    TCLAP::ValueArg<std::string> fileYArg ("y", "results", 
	    "File containing the training results", true, "", 
	    "file name or path", cmd);
	    
	cmd.parse( argc, argv );
	    options.numberOfLayers          = numLayersArg.getValue();
	    options.layerSizes              = splitToInts(layersArg.getValue(),',');
	    options.activationFunction      = activationFunctionArg.getValue();
	    options.samplesFile             = fileXArg.getValue();
	    options.resultsFile             = fileYArg.getValue();
        options.numberOfTrainingSamples = numTrainingSamplesArg.getValue();
        
    return options;
}

std::vector<int> &splitToInts(const std::string &s, char delim, 
    std::vector<int> &elems) 
{
    std::stringstream ss(s);
    std::string item;
    while (std::getline(ss, item, delim)) {
        elems.push_back(atoi(item.c_str()));
    }
    return elems;
}

std::vector<int> splitToInts(const std::string &s, char delim) 
{
    std::vector<int> elems;
    splitToInts(s, delim, elems);
    return elems;
}

void readCsvIntoMatrix(const std::string fileName, float* M, const int rows, 
                        const int columns)
{
    std::ifstream ifs (fileName.c_str());
	char dummy;
	float x;
	
	for (int i = 0; i < rows; ++i){
		for (int j = 0; j < columns; ++j){
			ifs >> x;
			M[i * columns + j] = x; 
			if (j < (columns - 1)) //ignore commas
				ifs >> dummy;
		}
	}
}

void readResultsIntoMatrix(const std::string fileName, float* M, const int rows, 
                            const int columns)
{
    std::ifstream ifs (fileName.c_str());
	int x;
	
	for (int i = 0; i < rows; ++i){
			ifs >> x;
			for (int j = 0; j < columns; j++)
			    M[i * columns + j] = 0.0;
			M[i * columns + x - 1] = 1.0; 
	}
}

void GPU_fill_rand(float *A, int nr_rows_A, int nr_cols_A) {
    printf("Fill rand: (%p,%d, %d)\n", A, nr_rows_A, nr_cols_A);
    // Create a pseudo-random number generator
    hiprandGenerator_t prng;
    hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);

    // Set the seed for the random number generator using the system clock
    hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long) clock());

    // Fill the array with random numbers on the device
    hiprandGenerateUniform(prng, A, nr_rows_A * nr_cols_A);
}

void printMatrix(float *M, int rows, int columns)
{
    printf("M:\n");
    for (int i = 0; i < rows; i++){
	    for (int j = 0; j < columns; j++)
	        printf("%f, ", M[i * columns + j]);
	    printf("\n");
	} 
}
//==============================================================================
// KERNELS WRAPPER FUNCTIONS
//==============================================================================
template<typename MapFunction,
         typename ReduceFunction>
float ZipMapReduce(float* d_X, float* d_Y, int size, MapFunction mapFunction, 
                   float neutralElement, ReduceFunction reduceFunction)
{
    float *R, *d_R, r = neutralElement;
    dim3 dimBlock(TILE_DIM);
    dim3 dimGrid((size + dimBlock.x - 1) / dimBlock.x);
    
    //Create auxiliary vector
    R = (float *) malloc (sizeof(float) * dimGrid.x);
    hipError_t err = hipMalloc((void **)&d_R, sizeof(float) * dimGrid.x);

    //Reduce to vector R
	ZipMapReduceKernel<<<dimGrid, dimBlock>>>(d_X, d_Y, d_R, size, mapFunction, neutralElement, reduceFunction);
    hipDeviceSynchronize();
    hipMemcpy(R, d_R, dimGrid.x * sizeof(float), hipMemcpyDeviceToHost);
    /*printf("Reduced to %d values: \n", dimGrid.x);
    for (int i = 0; i < dimGrid.x; i++)
    {
        printf("%f, ", R[i]);
    }
    printf("\n");*/
    //Reduce remaining values in host
    for (int i = 0; i < dimGrid.x; i++)
        r = reduceFunction(r, R[i]);
    hipFree(d_R);
    free(R);
    return r;
}
//==============================================================================
// KERNELS
//==============================================================================
template<typename UnaryFunction>
__global__ void MatMul(float* A, float* B, float* C, int ARows, int ACols, 
    int BRows, int BCols, int CRows, int CCols, bool addBias, UnaryFunction activationFunction ) 
{
    float CValue = 0;
    int Row = blockIdx.y * TILE_DIM + threadIdx.y;
    int Col = blockIdx.x * TILE_DIM + threadIdx.x;
    int biasOffset = addBias ? 1 : 0; 
	
    __shared__ float As[TILE_DIM][TILE_DIM];
    __shared__ float Bs[TILE_DIM][TILE_DIM];

    for (int k = 0; k < (TILE_DIM + ACols - 1)/TILE_DIM; k++)           //floor(ACols/TILE_DIM)
    {
         if (k * TILE_DIM + threadIdx.x < ACols && Row < ARows)   
            As[threadIdx.y][threadIdx.x] = A[Row * ACols + k * TILE_DIM + threadIdx.x];
         else                                                 
            As[threadIdx.y][threadIdx.x] = 0.0;

         if (k * TILE_DIM + threadIdx.y < BRows && Col < BCols)   
            Bs[threadIdx.y][threadIdx.x] = B[(k * TILE_DIM + threadIdx.y + biasOffset) * BCols + Col]; //+1 one row if bias
         else      
            Bs[threadIdx.y][threadIdx.x] = 0.0;

         __syncthreads();

         for (int n = 0; n < TILE_DIM; ++n) 
            CValue += As[threadIdx.y][n] * Bs[n][threadIdx.x];

         __syncthreads();
    }
    if (addBias)
	{
		__shared__ float BiasRow[TILE_DIM];
	    
		if (threadIdx.y == 0){
		  if (Col < BCols){
			BiasRow[threadIdx.x] = B[Col];
		  }else{
		  	BiasRow[threadIdx.x] = 0.0;
			}
			}
	    __syncthreads();
		
		CValue += BiasRow[threadIdx.x];
		
		__syncthreads();
		
	}
	
    if (Row < CRows && Col < CCols) 
        C[((blockIdx.y * blockDim.y + threadIdx.y) * CCols) + 
          (blockIdx.x * blockDim.x) + threadIdx.x] =
            activationFunction(CValue);
}

template<typename MapFunction,
         typename ReduceFunction>
__global__ void ZipMapReduceKernel(float* X, float* Y, float* R, int size, 
                                    MapFunction mapFunction, float neutralElement, 
                                    ReduceFunction reduceFunction)
{
    __shared__ float sX[TILE_DIM];
    __shared__ float sY[TILE_DIM];
    __shared__ float sR[TILE_DIM];
  
    unsigned int i = blockIdx.x * TILE_DIM + threadIdx.x;
    unsigned int tid = threadIdx.x;
    unsigned int bid = blockIdx.x;
        
    //Load data from memory to shared memory collectively
    sX[tid] = X[i];
    sY[tid] = Y[i];
    sR[tid] = neutralElement;
    __syncthreads();
    
    //Zip and Map: sR <- Map(Zip(sX,sY))
    if (i < size) 
        sR[tid] = mapFunction(sX[tid],sY[tid]);
    __syncthreads();
    
    //Reduce
    for(unsigned int s = TILE_DIM / 2; s > 0; s >>= 1)
    {
        if (tid < s)
            sR[tid] = reduceFunction(sR[tid], sR[tid + s]);
        __syncthreads();
    }
    
    //Delegate (thread 0) writes to memory
    if (tid == 0)
        R[bid] = sR[0];
}
